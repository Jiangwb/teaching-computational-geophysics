/* 
 
 helloWorld example for CUDA
 
 compile with:
 
 > nvcc -arch=sm_20 hello_cuda.cu
 
 run with:
 
 > ./a.out
 
*/

#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

#define N 10

// cuda kernel (runs on GPU)

__global__ void sum_kernel(float* A,float* B, float* C, float* sum, int nmax)
{

  // thread id      
  int id = blockIdx.x*blockDim.x + threadIdx.x;
  
  // sums values
  if( id < nmax ){
    C[id] = A[id] + B[id];
    
    // no atomic summation -> wrong results...
    //*sum += C[id];

    // atomic operation to avoid race-conditions
    atomicAdd(sum,C[id]);    
    
  }
}

// main program (runs on CPU)

int main(void)
{

  float *A, *B, *C;
  float *A_d, *B_d, *C_d;

  float sum;
  float *sum_d;

  printf("hello CUDA: \n");
  
  // array on CPU (host)
  A = (float *) malloc(N*sizeof(float));
  B = (float *) malloc(N*sizeof(float));
  C = (float *) malloc(N*sizeof(float));
  
  // initializes
  for(int i=0; i<N; i++){
    A[i] = 1.0;
    B[i] = 2.0;
    C[i] = 0.0;
  }
  
  sum = 0.0f;
  
  // GPU preparation:
  
  // array on GPU (device)
  hipMalloc((void **) &A_d, N*sizeof(float));
  hipMalloc((void **) &B_d, N*sizeof(float));
  hipMalloc((void **) &C_d, N*sizeof(float));
  hipMalloc((void **) &sum_d, sizeof(float));
  
  // initializes on GPU with zero
  hipMemset(sum_d,0,sizeof(float));
  
  // copies arrays from CPU to GPU
  hipMemcpy(A_d,A,N*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(B_d,B,N*sizeof(float),hipMemcpyHostToDevice);
  
  // cuda kernel dimensions ( 3 blocks x 4 threads )
  int blocksize = 4;
  int nblock = N/blocksize+(N%blocksize==0?0:1);

  // launches cuda kernel  
  sum_kernel<<<nblock,blocksize>>>(A_d,B_d,C_d,sum_d,N);
  
  // copies back from GPU to CPU
  hipMemcpy(C,C_d,N*sizeof(float),hipMemcpyDeviceToHost);
  hipMemcpy(&sum,sum_d,sizeof(float),hipMemcpyDeviceToHost);
  
  // user output
  printf("result: \n");  
  for(int i=0;i<N;i++){
    printf("  %f \n",C[i]);
  }  
  printf("\n");
  printf("  sum = %f\n\n",sum);
  
  // releases memory on CPU
  free(A);
  free(B);
  free(C);
  
  // releases memory on GPU
  hipFree(A_d);   
  hipFree(B_d);   
  hipFree(C_d);   
  hipFree(sum_d);
}
